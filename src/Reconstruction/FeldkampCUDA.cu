#include "hip/hip_runtime.h"
#define LIBCBCT_API_EXPORT
#include "FeldkampCUDA.h"

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>
#include <hipfft/hipfft.h>

#define _USE_MATH_DEFINES
#include <math.h>

#include "Utils/CudaUtils.h"
#include "Utils/ImageUtils.h"
#include "Common/ProgressBar.h"

#define BLOCK_SIZE 8

__global__ void rampFilterKernel(vec2f *buffer, int detWidth, int detHeight, RampFilter filter) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= 0 && y >= 0 && x < detWidth && y < detHeight) {
        const float q = min(x, detWidth - x) / (0.5f * detWidth);
        if (filter == RampFilter::RamLak) {
            buffer[y * detWidth + x] *= fabsf(q);
        } else if (filter == RampFilter::SheppLogan) {
            buffer[y * detWidth + x] *= (2.0f / M_PI) * fabsf(sinf(0.5f * M_PI * q));
        } else {
            asm("trap;");
        }
    }
}

__global__ void sliceBackProjectionKernel(int z, int i, Geometry &geometry, float *slice, vec3i volSize, float *image,
                                          vec3i sinoSize) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= 0 && y >= 0 && x < volSize.x && y < volSize.y) {
        const float theta = 2.0f * (float)M_PI * i / sinoSize.z;
        const vec3f uvw = vox2pix(vec3i(x, y, z), theta, geometry);
        if (uvw.x >= 0 && uvw.y >= 0 && uvw.x < sinoSize.x && uvw.y < sinoSize.y) {
            const float val = bilerp(image, sinoSize.x, sinoSize.y, uvw.x - 0.5f, uvw.y - 0.5f);
            slice[y * volSize.x + x] += val * uvw.z / sinoSize.z;
        }
    }
}

void FeldkampCUDA::reconstruct(const FloatVolume &sinogram, FloatVolume &tomogram, const Geometry &geometry) const {
    const int detWidth = sinogram.size<0>();
    const int detHeight = sinogram.size<1>();
    const int nProj = sinogram.size<2>();
    const vec3i sinoSize(detWidth, detHeight, nProj);
    const vec3i volSize = geometry.volSize;
    tomogram.resize(volSize.x, volSize.y, volSize.z);

    Geometry *devGeom;
    CUDA_CHECK(hipMalloc(&devGeom, sizeof(Geometry)));
    CUDA_CHECK(hipMemcpy(devGeom, &geometry, sizeof(Geometry), hipMemcpyHostToDevice));

    // Allocate CUDA memories
    float *devImg;
    vec2f *devFft;
    float *devVolume;
    CUDA_CHECK(hipMalloc(&devImg, sizeof(float) * detWidth * detHeight));
    CUDA_CHECK(hipMalloc(&devFft, sizeof(vec2f) * detWidth * detHeight));
    CUDA_CHECK(hipMalloc(&devVolume, sizeof(float) * (uint64_t)volSize.x * (uint64_t)volSize.y * (uint64_t)volSize.z));
    CUDA_CHECK(
        hipMemset(devVolume, 0, sizeof(float) * (uint64_t)volSize.x * (uint64_t)volSize.y * (uint64_t)volSize.z));

    // FFT parameters
    const int rank = 1;
    int n[] = { detWidth };
    int inembed[] = { 0 };
    int onembed[] = { 0 };
    const int istride = 1;
    const int ostride = 1;
    const int idist = detWidth;
    const int odist = detWidth;
    const int batch = detHeight;

    hipfftHandle planForward;
    CUDA_CHECK(
        hipfftPlanMany(&planForward, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch));

    hipfftHandle planInverse;
    CUDA_CHECK(
        hipfftPlanMany(&planInverse, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2R, batch));

    ProgressBar pbar(nProj);
    pbar.setDescription("RECON: ");
    for (int i = 0; i < nProj; i++) {
        const uint64_t ptrOffset = detWidth * detHeight * (uint64_t)i;
        float *const imgPtr = sinogram.ptr() + ptrOffset;
        CUDA_CHECK(hipMemcpy(devImg, imgPtr, sizeof(float) * detWidth * detHeight, hipMemcpyHostToDevice));

        //Filter
        CUDA_CHECK(hipfftExecR2C(planForward, devImg, (float2 *)devFft));
        CUDA_SYNC_CHECK();

        const int nGridX = (detWidth + BLOCK_SIZE - 1) / BLOCK_SIZE;
        const int nGridY = (detHeight + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 grid(nGridX, nGridY, 1);
        rampFilterKernel<<<grid, block>>>(devFft, detWidth, detHeight, filter);
        CUDA_SYNC_CHECK();

        CUDA_CHECK(hipfftExecC2R(planInverse, (float2 *)devFft, devImg));
        CUDA_SYNC_CHECK();

        // Back projection
        for (int z = 0; z < volSize.z; z++) {
            float *const devSlice = devVolume + (uint64_t)volSize.x * (uint64_t)volSize.y * (uint64_t)z;
            const int nGridX = (volSize.x + BLOCK_SIZE - 1) / BLOCK_SIZE;
            const int nGridY = (volSize.y + BLOCK_SIZE - 1) / BLOCK_SIZE;
            dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
            dim3 grid(nGridX, nGridY, 1);
            sliceBackProjectionKernel<<<grid, block>>>(z, i, *devGeom, devSlice, volSize, devImg, sinoSize);
            CUDA_SYNC_CHECK();
        }
        CUDA_SYNC_CHECK();

        pbar.step();
    }

    CUDA_CHECK(hipMemcpy(tomogram.ptr(), devVolume, sizeof(float) * volSize.x * volSize.y * volSize.z,
                          hipMemcpyDeviceToHost));

    // Free CUDA memories
    CUDA_CHECK(hipFree(devImg));
    CUDA_CHECK(hipFree(devFft));
    CUDA_CHECK(hipFree(devVolume));
    CUDA_CHECK(hipFree(devGeom));
    CUDA_CHECK(hipfftDestroy(planForward));
    CUDA_CHECK(hipfftDestroy(planInverse));
    CUDA_SYNC_CHECK();
}
